#include "hip/hip_runtime.h"
//*Side_Colision_Force(p);
//Top_Colision_Force(p);
//Floor_Colision_Force(p,0.90f);
//Ball_Colision_Force(p,1.0f);
//ParticleDerivative(p,temp1,mask);
//ScaleVector(temp1,DeltaT,p);*/
//
//*void Floor_Colision_Force(ParticleSystem *p,float loss_param){
//
//	for(int i=0;i<p->n;i++){
//		if(p->p[i].x[1]-p->p[i].R< -20.0f && p->p[i].v[1]<-0.2f){
//			p->p[i].v[1]= -p->p[i].v[1]*loss_param;
//			p->p[i].x[1]=-20.0f+p->p[i].R;
//		}
//	}
//}
//
//void Top_Colision_Force(ParticleSystem *sys){
//		for(int i=0;i<sys->n;i++){
//			if(sys->p[i].x[1]+sys->p[i].R> 20.0f){
//				sys->p[i].v[1]= -sys->p[i].v[1];
//				sys->p[i].x[1]= 20.0f - sys->p[i].R;
//			}
//	}
//
//}
//
//void Side_Colision_Force(ParticleSystem *sys){
//	for(int i=0;i<sys->n;i++){
//		if(sys->p[i].x[0]+sys->p[i].R> 20.0f){
//			sys->p[i].v[0]= -sys->p[i].v[0];
//			sys->p[i].x[0]= 20.0f-sys->p[i].R;
//		}
//		if(sys->p[i].x[0]-sys->p[i].R< -20.0f){
//			sys->p[i].v[0]= -sys->p[i].v[0];
//			sys->p[i].x[0]= -20.0f+sys->p[i].R;
//		}
//	}
//}
//
//void Gravity_Force(ParticleSystem *p,float gravity){
//	for(int i=0;i<p->n;i++){
//		p->p[i].f[0] += 0.0f;
//		p->p[i].f[1] += - (p->p[i].m*gravity);
//	}
//}*/

#include "generalCuda.h"
__device__ inline int getutid()
{
int threadsPerBlock = blockDim.x * blockDim.y;
int tidWithinBlock = threadIdx.x + threadIdx.y * blockDim.x;
int gid = blockIdx.x + blockIdx.y * gridDim.x;
return gid * threadsPerBlock + tidWithinBlock;
}

int ParticleGetStateAll(ParticleSystem *p, float *dst){
	int i;
	for(i=0; i < p->n; i++){
		*(dst++) = p->p[i].x[0];
		*(dst++) = p->p[i].x[1];
		*(dst++) = p->p[i].v[0];
		*(dst++) = p->p[i].v[1];
		*(dst++) = p->p[i].f[0];
		*(dst++) = p->p[i].f[1];
		*(dst++) = p->p[i].m;
		*(dst++) = p->p[i].R;
	}
	return i;
}

///* scatter state from src into the particles */
void ParticleSetStateAll(ParticleSystem *p, float *src){
int i;
	for(i=0; i < p->n; i++){
		p->p[i].x[0] = *(src++);
		p->p[i].x[1] = *(src++);
		p->p[i].v[0] = *(src++);
		p->p[i].v[1] = *(src++);
		p->p[i].f[0] = *(src++);
		p->p[i].f[1] = *(src++);
		p->p[i].m = *(src++);
		p->p[i].R = *(src++);
	}
}

__global__ void gravityForceKernel(float *d_system,float gravity,int no){
	int thIndex = getutid();

	int d_system_index = thIndex*8;
	if(thIndex<no){
		d_system[d_system_index+5] += - (d_system[d_system_index+6]*gravity);
	}
}
extern "C" void callGravityForceKernel(float *d_system,ParticleSystem *sys, float gravity){
	dim3 dimBlock(16,16);
	dim3 dimGrid(32,32);
	if(sys->n > dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y){
		printf("\n to many objects. limit is: %d",dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y);
		return;
	}
	gravityForceKernel<<<dimGrid,dimBlock>>>(d_system,gravity,sys->n);
}
/*
void Top_Colision_Force(ParticleSystem *sys,float loss_param){
		for(int i=0;i<sys->n;i++){
			if(sys->p[i].x[1]+sys->p[i].R> 20.0f){
				sys->p[i].v[1]= -sys->p[i].v[1]*loss_param;
				sys->p[i].x[1]= 20.0f - sys->p[i].R;
			}
	}

}
*/
__global__ void topColisionKernel(float *d_system,float loss_param,int no,float maxY){
	int thIndex = getutid();

	int d_system_index = thIndex*8;
	if(thIndex<no){
		if(d_system[d_system_index+1] + d_system[d_system_index+7]> maxY){
			d_system[d_system_index+3]= -d_system[d_system_index+3]*loss_param;
			d_system[d_system_index+1]= maxY-d_system[d_system_index+7];
		}
	}
}
extern "C" void callTopColisionKernel(float *d_system,float *h_system, ParticleSystem *sys, float loss_param, float maxY){
	dim3 dimBlock(16,16);
	dim3 dimGrid(32,32);
	if(sys->n > dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y){
		printf("\n to many objects. limit is: %d",dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y);
		return;
	}
	topColisionKernel<<<dimGrid,dimBlock>>>(d_system,loss_param,sys->n ,maxY);

}


__global__ void sideColisionKernel(float *d_system,float loss_param,int no,float maxX){
	int thIndex = getutid();

	int d_system_index = thIndex*8;
	if(thIndex<no){
		if(d_system[d_system_index] + d_system[d_system_index+7]> maxX){
			d_system[d_system_index+2]= -d_system[d_system_index+2]*loss_param;
			d_system[d_system_index]= maxX-d_system[d_system_index+7];
		}
		if(d_system[d_system_index] - d_system[d_system_index+7]< -maxX){
			d_system[d_system_index+2]= -d_system[d_system_index+2]*loss_param;
			d_system[d_system_index]= -maxX+d_system[d_system_index+7];
		}
		if(abs(d_system[d_system_index+2])<=3.0f && d_system[d_system_index+4]==0.00f) d_system[d_system_index+2]=d_system[d_system_index+2]*(-0.1f);
	}
}

extern "C" void callSideColisionKernel(float *d_system,float *h_system, ParticleSystem *sys, float loss_param, float maxX){
	dim3 dimBlock(16,16);
	dim3 dimGrid(32,32);
	if(sys->n > dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y){
		printf("\n to many objects. limit is: %d",dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y);
		return;
	}
	sideColisionKernel<<<dimGrid,dimBlock>>>(d_system,loss_param,sys->n ,maxX);
}
/*
	if(p->p[i].x[1]-p->p[i].R< -20.0f){
			p->p[i].x[1]=-20.0f+p->p[i].R;
			if(abs(p->p[i].v[1]) >3.0f) p->p[i].v[1]= -p->p[i].v[1]*loss_param;
			else
				p->p[i].f[1]=0.0f;
		}
*/
__global__ void floorColisionKernel(float *d_system,float loss_param,int no,float minY){
	int thIndex = getutid();

	int d_system_index = thIndex*8;
	if(thIndex<no){
		if(d_system[d_system_index+1] - d_system[d_system_index+7]< minY){
			d_system[d_system_index+1]=minY+d_system[d_system_index+7];
			if(abs(d_system[d_system_index+3])>3.0f) d_system[d_system_index+3]= -d_system[d_system_index+3]*loss_param;
			else
			d_system[d_system_index+5]= 0.0f;
		}
	}
}
extern "C" void callFloorColisionKernel(float *d_system,float *h_system, ParticleSystem *sys, float loss_param, float minY){
	dim3 dimBlock(16,16);
	dim3 dimGrid(32,32);
	if(sys->n > dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y){
		printf("\n to many objects. limit is: %d",dimBlock.x*dimBlock.y*dimGrid.x*dimGrid.y);
		return;
	}
	floorColisionKernel<<<dimGrid,dimBlock>>>(d_system,loss_param,sys->n ,minY);
	

}
extern "C" void callMallocSystemMemory(ParticleSystem *sys,float **h_system, float **d_system){
	float *d_temp =NULL;
	hipMalloc((void**) d_system,sizeof(float)* 8*sys->n);
	//hipMalloc((void**) &d_temp,sizeof(float)* 8*sys->n);
	hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        printf( "Cuda error: %s: %s.\n", "error", 
                                  hipGetErrorString( err) );
    } 
	//float *temp= (float *)malloc(sizeof(float)* 8*sys->n);
	*h_system = (float *)malloc(sizeof(float)* 8*sys->n);
	//hipFree(d_temp);
	//free(temp);
	
//minY = -20.0f
	//floorColisionKernel<<<>>>(d_system,loss_param,sys->n);
}
extern "C" void callClearMemory(float *d_system, float *h_system){
	hipFree(d_system);
	free(h_system);
}
extern "C" void uploadMemorySystem(ParticleSystem *sys, float *h_system, float *d_system){
	ParticleGetStateAll(sys,h_system);
	hipMemcpy(d_system,h_system,sizeof(float)* 8*sys->n,hipMemcpyHostToDevice);
}
extern "C" void downloadMemorySystem(ParticleSystem *sys, float *h_system, float *d_system){
	hipMemcpy(h_system,d_system,sizeof(float)* 8*sys->n,hipMemcpyDeviceToHost);
	ParticleSetStateAll(sys,h_system);
}